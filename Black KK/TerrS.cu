#include "hip/hip_runtime.h"
#include"TerrS.cuh"
#include"ScreenDraw.h"
#include<Windows.h>
#include<iostream>
__constant__ int d_perm[512];
__constant__ float d_quan[256];
__global__ void perlinNoiseKernel(float* noise, int width, float freq) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= width) 
        return;
    float xf = x / freq;
    int xi0 = (static_cast<int>(xf) & 255);
    int xi1 = (xi0 + 1) & 255;
    float tx = xf - static_cast<int>(xf);
    float u = tx * tx * tx * (tx * (tx * 6 - 15) + 10);
    int FirstIndex = d_perm[xi0];
    int SecondIndex = d_perm[xi1];
    float noisenum = d_quan[FirstIndex];
    float Subnum = d_quan[SecondIndex] - d_quan[FirstIndex];
    noisenum += Subnum * u;
    noise[x] = noisenum;
}
void TerrS::initThep()
{
    const int PerSize = 256;
    std::vector<int> perm(PerSize);
    float gradients[PerSize];
    std::srand(std::time(0));
    for (int i = 0; i < PerSize;++i) {
        perm[i] = i % PerSize;
    }
    int UpMax = 850;
    int LowDown = 400;
    int countnum1 = 0;
    int countnum2 = 0;
    bool Can1 = true, Can2 = true;
    int TheMedium = Width / Freq;
    TheMedium /= 2;
    for (int medium=TheMedium,medium2=TheMedium+1; Can1||Can2;--medium,++medium2) 
    {
        if (medium < 0)
            Can1 = false;
        if (medium2 >= PerSize)
            Can2 = false;
        if(countnum1>5)
        {
            countnum1 = 0;
            if(LowDown>=100)
                LowDown -= 100;
        }
        if(countnum2>11)
        {
            countnum2 = 0;
            if (UpMax > 100)
                UpMax -= 100;
        }
        int TheSub = UpMax - LowDown;
        if (Can1)
        {
            float RandNum = rand() % TheSub + LowDown;
            RandNum /= 1000.0;
            gradients[medium] = RandNum;
        }
        if (Can2)
        {
            float RandNum2 = rand() % TheSub + LowDown;
            RandNum2 /= 1000.0;
            gradients[medium2] = RandNum2;
        }
        ++countnum1;
        ++countnum2;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(d_perm), perm.data(), 2 * PerSize * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_quan), gradients, PerSize * sizeof(float));
}
void TerrS::SpawnTerr(std::vector<float> vectorNoise)
{
    int PanDuan = 0;//�����жϴ����߻��Ǵ��м���ɢ��0Ϊ�м䣬1Ϊ����
    int CurrentLie1 = Width / 2;
    int CurrentLie2 = CurrentLie1 + 1;
    float HeightTemp = Height / 4;
    int Hang1 = HeightTemp *2;
    int Hang2 = HeightTemp * 3;
    int HangSum = Hang2 - Hang1;
    bool Can1 = true, Can2 = true;
    int countnum1 = 0;
    while(Can1||Can2)
    {
       if(PanDuan==0)
       {
           if (countnum1 > Width/10) 
           {
               countnum1 = 0;
               PanDuan =(PanDuan+1)%3;
               continue;
           }
           if (Can1)
           {
               int RandHang =0;
               int IncreaseWay = 0;
               int useTemp = rand() % 4;
               if(useTemp<=1)
               {
                   RandHang = Hang1 - rand() % HangSum/4*3;
                   IncreaseWay = 1;
               }
               else
               {
                   RandHang=Hang2- rand() % HangSum;
                   IncreaseWay = -1;
               }
               float Temp = HeightTemp * vectorNoise[CurrentLie1];
               int MaxBarr1 = static_cast<int>(Temp);
               for (int i1 = RandHang;i1>=0&&i1<Height&&MaxBarr1>0;i1+=IncreaseWay)
               {
                   (*TheTerrS)[i1][CurrentLie1] = 1;
                   MaxBarr1 -= 1;
               }
           }
           if(Can2)
           {
               int RandHang = 0;
               int IncreaseWay = 0;
               int useTemp = rand() % 4;
               if (useTemp <= 1)
               {
                   RandHang = Hang1 - rand() % HangSum/4*3;
                   IncreaseWay = 1;
               }
               else
               {
                   RandHang = Hang2 - rand() % HangSum;
                   IncreaseWay = -1;
               }
               float Temp = HeightTemp * vectorNoise[CurrentLie2];
               int MaxBarr2 = static_cast<int>(Temp);
               for (int i1 = RandHang; i1>=0&&i1<Height&&MaxBarr2>0;i1+=IncreaseWay)
               {
                   (*TheTerrS)[i1][CurrentLie2] = 1;
                   MaxBarr2 -= 1;
               }
           }
       }
       else if(PanDuan==1)
       {
           if (countnum1 > Width/8)
           {
               countnum1 = 0;
               PanDuan=(PanDuan + 1) % 3;
               continue;
           }
           if(Can1)
           {
               int RandPanduan = rand() % 4;
               int RanHang = 0;
               int IncreaseWay = 0;
               if (RandPanduan <= 1)
               {
                   RanHang = 0;
                   IncreaseWay = 1;
               }
               else
               {
                   RanHang = rand() % HangSum + Hang2;
                   IncreaseWay = 1;
               }
               float Temp = HeightTemp * vectorNoise[CurrentLie1];
               int MaxBarr1 = static_cast<int>(Temp);
               for(int i1=RanHang;i1>=0&&i1<Height&&MaxBarr1>0;i1+=IncreaseWay)
               {
                   (*TheTerrS)[i1][CurrentLie1] = 1;
                   --MaxBarr1;
               }
           }
           if(Can2)
           {
               int RandPanduan = rand() % 4;
               int RanHang = 0;
               int IncreaseWay = 0;
               if (RandPanduan <= 1)
               {
                   RanHang = rand() % HangSum;
                   IncreaseWay = 1;
               }
               else
               {
                   RanHang = rand() % HangSum + Hang2;
                   IncreaseWay = 1;
               }
               float Temp = HeightTemp * vectorNoise[CurrentLie2];
               int MaxBarr2 = static_cast<int>(Temp);
               for (int i1 = RanHang; i1 >= 0 && i1 < Height && MaxBarr2>0; i1 += IncreaseWay)
               {
                   (*TheTerrS)[i1][CurrentLie2] = 1;
                   --MaxBarr2;
               }
           }
       }
       else 
       {
           if (countnum1 > Width / 15)
           {
               countnum1 = 0;
               PanDuan = (PanDuan + 1) % 3;
               continue;
           }
       }
       --CurrentLie1;
       if (CurrentLie1 < 0)
           Can1 = false;
       ++CurrentLie2;
       if (CurrentLie2 >= Width)
           Can2 = false;
       ++countnum1;
    }
}
TerrS::TerrS(int wid,int hei):Width(wid),Height(hei),Freq(5)
{
    TheTerrS = std::make_shared<std::vector<std::vector<int>>>(Height, std::vector<int>(Width, 0));
    NoiseNum = std::vector<float>(Width, 0.0);
    float* d_noise;
    hipMalloc(&d_noise, Width*sizeof(float));
    initThep();
    int threadsPerBlock = 64;
    int numBlocks = (Width + threadsPerBlock - 1) / threadsPerBlock;
    perlinNoiseKernel << <numBlocks, threadsPerBlock>> > (d_noise, Width,Freq); 
        float* Thenoise = new float[Width];
    hipMemcpy(Thenoise, d_noise, Width*sizeof(float), hipMemcpyDeviceToHost);
    {
        std::vector<float> vectorNoise;
        for (int i = 0; i < Width; ++i)
        {
            vectorNoise.push_back(Thenoise[i]);
        }
        SpawnTerr(vectorNoise);
    }
    for(int i=0;i<Height;++i)
    {
        (*TheTerrS)[i][0] = 0;
        (*TheTerrS)[i][Width - 1] = 0;
    }
    delete []Thenoise;
    hipFree(d_noise);
}

TerrS::~TerrS()
{
}


