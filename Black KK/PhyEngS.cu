#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "PhyEngS.cuh"
#include "physis.h"
#include"Boss.h"
#include"Player.h"
#include"AINode.h"
#include<chrono>
#include<fstream>
#include<thread>
__global__ void UpdateBulletKernel(int* speedx, int* speedy, int* NowX, int* NowY, int* IfBossBulletValid, 
	int* D_TheGrid, int TheWidth, int TheHeight, int currentIndex,int PlayerX,int PlayerY,int*BossHitPlayer,int*PlayerNowX,int *PlayerNowY,int *IfPlayerBulletValid
    ,int maxPlayerIndex)
{
	__shared__ int ThePlayerX;
	ThePlayerX = PlayerX;
	__shared__ int ThePlayerY;
	ThePlayerY = PlayerY;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= currentIndex)
		return;
	if (IfBossBulletValid[idx]==1)
	{
	    NowX[idx] += speedx[idx];
	    NowY[idx] += speedy[idx];
		if (NowX[idx] < 0 || NowX[idx] >= TheWidth || NowY[idx] < 0 || NowY[idx] >= TheHeight)
		{
			IfBossBulletValid[idx] = 0;
			return;
		}
		if (NowX[idx] == ThePlayerX && NowY[idx] == ThePlayerY) {
			BossHitPlayer[idx] = 1;
			IfBossBulletValid[idx] = 0;
			return;
		}
		if (D_TheGrid[NowY[idx] * TheWidth + NowX[idx]] == 1)
		{
			IfBossBulletValid[idx] = 0;
			return;
		}
		for(int i=0;i<maxPlayerIndex;++i)
		{
			if (IfPlayerBulletValid[i] == 1 && NowX[idx] == PlayerNowX[i] && NowY[idx] == PlayerNowY[i])
			{
				IfBossBulletValid[idx] = 0;
				return;
		    }
		}
	}
}
__global__ void UpdatePlayerBulletKernel(int* aspeedx, int* aspeedy, int* aNowX, int* aNowY, int* IfPlayerBulletValid, 
	int* D_TheGrid, int TheWidth, int TheHeight, int currentIndex,int BossX,int BossY,int *PlayerHitBoss)
{
	__shared__ int TheBossX;
	TheBossX = BossX;
	__shared__ int TheBossY;
	TheBossY = BossY;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= currentIndex)
		return;
	if (IfPlayerBulletValid[idx]==1)
	{
		aNowX[idx] += aspeedx[idx];
		aNowY[idx] += aspeedy[idx];
		if (aNowX[idx] < 0 || aNowX[idx] >= TheWidth || aNowY[idx] < 0 || aNowY[idx] >= TheHeight)
		{
			IfPlayerBulletValid[idx] = 0;
			return;
		}
        if(aNowX[idx]==TheBossX&&aNowY[idx]==TheBossY)
		{
			IfPlayerBulletValid[idx] = 0;
			PlayerHitBoss[idx] = 1;
		}
		if (D_TheGrid[aNowY[idx] * TheWidth + aNowX[idx]] == 1)
			IfPlayerBulletValid[idx] = 0;
	}
}
void PhyEngS::PrePrepare()
{
	for (auto& i : PlayerHitBoss)
	{
		i = 0;
	}
	for (auto& i : BossHitPlayer)
	{
		i = 0;
	}
	for (int i = 0; i < maxBossIndex; ++i)
	{
		if (IfBossBulletValid[i] == 1)
			TheScreenDraw->Draw(NowX[i], NowY[i], ' ');
	}
	for (int i = 0; i < maxPlayerIndex; ++i)
	{
		if (IfPlayerBulletValid[i] == 1)
			TheScreenDraw->Draw(NowXPlayer[i], NowYPlayer[i], ' ');
	}
}
void PhyEngS::AfterCollision()
{
	abBoss->BossBeHitted();
	abPlayer->PlayerBeHitted();
}
PhyEngS::PhyEngS(int aPlayerX, int aPlayerY, int aBossX, int aBossY)
	:maxBossIndex(256), maxPlayerIndex(64), 
	currentBossIndex(0), currentPlayerIndex(0), BossX(aBossX), BossY(aBossY), PlayerX(aPlayerX), PlayerY(aPlayerY)
{
	TheScreenDraw = std::make_shared<MyScreenDraw>();
	TheTerr = std::make_shared<TerrS>(TheScreenDraw->GetWidth(), TheScreenDraw->GetHeight());
	PlayerHitBoss.resize(maxPlayerIndex);
	BossHitPlayer.resize(maxBossIndex);
	speedx.resize(maxBossIndex);
	speedy.resize(maxBossIndex);
	NowX.resize(maxBossIndex);
	NowY.resize(maxBossIndex);
	IfBossBulletValid.resize(maxBossIndex);
	IfPlayerBulletValid.resize(maxPlayerIndex);
	speedxPlayer.resize(maxPlayerIndex);
	speedyPlayer.resize(maxPlayerIndex);
	NowXPlayer.resize(maxPlayerIndex);
	NowYPlayer.resize(maxPlayerIndex);
	for (int i = 0; i < maxBossIndex; ++i)
	{
		IfBossBulletValid[i] = 0;
	}
	for (int i = 0; i < maxPlayerIndex; ++i)
	{
		IfPlayerBulletValid[i] = 0;
	}
	int Width = TheTerr->GetWidth();
	int Height = TheTerr->GetHeight();
	int* TheGrid = new int[Width * Height];
	for (int i = 0; i < Height; ++i)
	{
		for (int j = 0; j < Width; ++j)
		{
			TheGrid[i * Width + j] = (*TheTerr->GetGrid())[i][j];
		}
	}
	for (int i = 0; i < Height; ++i) {
		for(int j=0;j<Width;++j)
		{
			if (!TheTerr->IfCanMove(j, i))
				TheScreenDraw->Draw(j, i, L'#', 0x0002);
		}
	}
	hipMalloc((void**)&D_TheGrid, Width * Height * sizeof(int));
	hipMemcpy(D_TheGrid, TheGrid, Width * Height * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&D_speedx, maxBossIndex * sizeof(int));
	hipMalloc((void**)&D_speedy, maxBossIndex * sizeof(int));
	hipMalloc((void**)&D_NowX, maxBossIndex * sizeof(int));
	hipMalloc((void**)&D_NowY, maxBossIndex * sizeof(int));
	hipMalloc((void**)&D_IfBossBulletValid, maxBossIndex * sizeof(int));
	hipMalloc((void**)&D_speedxPlayer, maxPlayerIndex * sizeof(int));
	hipMalloc((void**)&D_speedyPlayer, maxPlayerIndex * sizeof(int));
	hipMalloc((void**)&D_NowXPlayer, maxPlayerIndex * sizeof(int));
	hipMalloc((void**)&D_NowYPlayer, maxPlayerIndex * sizeof(int));
	hipMalloc((void**)&D_IfPlayerBulletValid, maxPlayerIndex * sizeof(int));
	hipMalloc((void**)&D_PlayerHitBoss, maxPlayerIndex * sizeof(int));
	hipMalloc((void**)&D_BossHitPlayer, maxBossIndex * sizeof(int));
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	hipStreamCreate(&stream3);
	delete[] TheGrid;
}
void PhyEngS::Innitialization(std::shared_ptr<Boss> aBoss, std::shared_ptr<Player> aPlayer)
{
	abBoss = aBoss;
	abPlayer = aPlayer;
}
PhyEngS::~PhyEngS()
{
	hipFree(D_TheGrid);
	hipFree(D_speedx);
	hipFree(D_speedy);
	hipFree(D_NowX);
	hipFree(D_NowY);
	hipFree(D_IfBossBulletValid);
	hipFree(D_speedxPlayer);
	hipFree(D_speedyPlayer);
	hipFree(D_NowXPlayer);
	hipFree(D_NowYPlayer);
	hipFree(D_IfPlayerBulletValid);
	hipFree(D_PlayerHitBoss);
	hipFree(D_BossHitPlayer);
	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);
	hipStreamDestroy(stream3);
}
void PhyEngS::UpDateBullet()
{
	PrePrepare();
	hipMemcpyAsync(D_speedx, speedx.data(), maxBossIndex * sizeof(int), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(D_speedxPlayer, speedxPlayer.data(), maxPlayerIndex * sizeof(int), hipMemcpyHostToDevice, stream2);
	hipMemcpyAsync(D_speedy, speedy.data(), maxBossIndex * sizeof(int), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(D_speedyPlayer, speedyPlayer.data(), maxPlayerIndex * sizeof(int), hipMemcpyHostToDevice, stream2);
	hipMemcpyAsync(D_NowX, NowX.data(), maxBossIndex * sizeof(int), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(D_NowXPlayer, NowXPlayer.data(), maxPlayerIndex * sizeof(int), hipMemcpyHostToDevice, stream2);
	hipMemcpyAsync(D_NowY, NowY.data(), maxBossIndex * sizeof(int), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(D_NowYPlayer, NowYPlayer.data(), maxPlayerIndex * sizeof(int), hipMemcpyHostToDevice, stream2);
	hipMemcpyAsync(D_IfBossBulletValid, IfBossBulletValid.data(), maxBossIndex * sizeof(int), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(D_IfPlayerBulletValid, IfPlayerBulletValid.data(), maxPlayerIndex * sizeof(int), hipMemcpyHostToDevice, stream2);
	hipMemcpyAsync(D_PlayerHitBoss, PlayerHitBoss.data(), maxBossIndex * sizeof(int), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(D_BossHitPlayer, BossHitPlayer.data(), maxPlayerIndex * sizeof(int), hipMemcpyHostToDevice, stream2);
	hipStreamSynchronize(stream1);
	hipStreamSynchronize(stream2);
	int bossBlockSize = 64;
	int bossNumBlocks = (maxBossIndex + bossBlockSize - 1) / bossBlockSize;
	int playerBlockSize = 64;
	int playerNumBlocks = (maxPlayerIndex + playerBlockSize - 1) / playerBlockSize;
	UpdateBulletKernel << <bossNumBlocks, bossBlockSize, 0, stream3 >> > (D_speedx, D_speedy, D_NowX, D_NowY, D_IfBossBulletValid,
		D_TheGrid, TheTerr->GetWidth(), TheTerr->GetHeight(), maxBossIndex, PlayerX, PlayerY, D_BossHitPlayer,D_NowXPlayer,D_NowYPlayer,D_IfPlayerBulletValid,maxPlayerIndex);
	hipStreamSynchronize(stream3);
	UpdatePlayerBulletKernel << <playerNumBlocks, playerBlockSize, 0, stream3 >> > (D_speedxPlayer, D_speedyPlayer, D_NowXPlayer, D_NowYPlayer, D_IfPlayerBulletValid,
		D_TheGrid, TheTerr->GetWidth(), TheTerr->GetHeight(), maxPlayerIndex, BossX, BossY, D_PlayerHitBoss);
	hipStreamSynchronize(stream3);
	hipMemcpyAsync(speedx.data(), D_speedx, maxBossIndex * sizeof(int), hipMemcpyDeviceToHost, stream1);
	hipMemcpyAsync(speedxPlayer.data(), D_speedxPlayer, maxPlayerIndex * sizeof(int), hipMemcpyDeviceToHost, stream2);
	hipMemcpyAsync(speedy.data(), D_speedy, maxBossIndex * sizeof(int), hipMemcpyDeviceToHost, stream1);
	hipMemcpyAsync(speedyPlayer.data(), D_speedyPlayer, maxPlayerIndex * sizeof(int), hipMemcpyDeviceToHost, stream2);
	hipMemcpyAsync(NowX.data(), D_NowX, maxBossIndex * sizeof(int), hipMemcpyDeviceToHost, stream1);
	hipMemcpyAsync(NowXPlayer.data(), D_NowXPlayer, maxPlayerIndex * sizeof(int), hipMemcpyDeviceToHost, stream2);
	hipMemcpyAsync(NowY.data(), D_NowY, maxBossIndex * sizeof(int), hipMemcpyDeviceToHost, stream1);
	hipMemcpyAsync(NowYPlayer.data(), D_NowYPlayer, maxPlayerIndex * sizeof(int), hipMemcpyDeviceToHost, stream2);
	hipMemcpyAsync(IfBossBulletValid.data(), D_IfBossBulletValid, maxBossIndex * sizeof(int), hipMemcpyDeviceToHost, stream1);
	hipMemcpyAsync(IfPlayerBulletValid.data(), D_IfPlayerBulletValid, maxPlayerIndex * sizeof(int), hipMemcpyDeviceToHost, stream2);
	hipMemcpyAsync(PlayerHitBoss.data(), D_PlayerHitBoss, maxBossIndex * sizeof(int), hipMemcpyDeviceToHost, stream1);
	hipMemcpyAsync(BossHitPlayer.data(), D_BossHitPlayer, maxPlayerIndex * sizeof(int), hipMemcpyDeviceToHost, stream2);
	hipStreamSynchronize(stream1);
	hipStreamSynchronize(stream2);
	AfterCollision();
	Tosy->Wait();
	Draw();
	TheScreenDraw->Render();
}

void PhyEngS::Draw()
{
	for (int i = 0; i < maxBossIndex; ++i)
	{
		if (IfBossBulletValid[i] == 1)
			TheScreenDraw->Draw(NowX[i], NowY[i], '*');
	}
	for (int i = 0; i < maxPlayerIndex; ++i)
	{
		if (IfPlayerBulletValid[i] == 1)
			TheScreenDraw->Draw(NowXPlayer[i], NowYPlayer[i], 'o');
	}
	TheScreenDraw->Draw(BossX, BossY, BossChar);
	TheScreenDraw->Draw(PlayerX, PlayerY, PlayerChar);
}
int PhyEngS::GetPlayerBeHittedTime()
{
	int j = 0;
	for(int i=0;i<maxBossIndex;++i)
	{
		if (BossHitPlayer[i] == 1)
			++j;
	}
	return j;
}
int PhyEngS::GetBossBeHittedTime()
{
	int j = 0;
	for (int i = 0; i < maxPlayerIndex; ++i)
	{
		if (PlayerHitBoss[i] == 1)
			++j;
	}
	return j;
}
