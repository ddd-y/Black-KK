#include "hip/hip_runtime.h"
#include "PhyEngS.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "physis.h"
#include"Boss.h"
#include"Player.h"
#include"AINode.h"
#include<chrono>
#include<fstream>
#include<thread>
__global__ void UpdateBulletKernel(int* speedx, int* speedy, int* NowX, int* NowY, int* IfBossBulletValid, 
	int* D_TheGrid, int TheWidth, int TheHeight, int currentIndex,int PlayerX,int PlayerY,int*BossHitPlayer)
{
	int ThePlayerX = PlayerX;
	int ThePlayerY = PlayerY;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= currentIndex)
		return;
	if (IfBossBulletValid[idx]==1)
	{
	    NowX[idx] += speedx[idx];
	    NowY[idx] += speedy[idx];
		if (NowX[idx] < 0 || NowX[idx] >= TheWidth || NowY[idx] < 0 || NowY[idx] >= TheHeight)
		{
			IfBossBulletValid[idx] = 0;
			return;
		}
		int isValid = (NowX[idx] != ThePlayerX || NowY[idx] != ThePlayerY);
		IfBossBulletValid[idx] *= isValid;
		if (!isValid)
			BossHitPlayer[idx] = 1;
		if (D_TheGrid[NowY[idx] * TheWidth + NowX[idx]] == 1)
			IfBossBulletValid[idx] = 0;
	}
}
__global__ void UpdatePlayerBulletKernel(int* aspeedx, int* aspeedy, int* aNowX, int* aNowY, int* IfPlayerBulletValid, 
	int* D_TheGrid, int TheWidth, int TheHeight, int currentIndex,int BossX,int BossY,int *PlayerHitBoss,int *BossNowX,int *BossNowY,int *IfBossBulletValid,int maxBossIndex)
{
	int TheBossX = BossX;
	int TheBossY = BossY;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= currentIndex)
		return;
	if (IfPlayerBulletValid[idx]==1)
	{
		int isValid = 1;
		for (int i = 0; i < maxBossIndex; ++i)
		{
			if (IfBossBulletValid[i] == 1 && aNowX[idx] == BossNowX[i] && aNowY[idx] == BossNowY[i])
			{
				IfPlayerBulletValid[idx] = 0;
				break;
			}
		}
		aNowX[idx] += aspeedx[idx];
		aNowY[idx] += aspeedy[idx];
		if (aNowX[idx] < 0 || aNowX[idx] >= TheWidth || aNowY[idx] < 0 || aNowY[idx] >= TheHeight)
		{
			IfPlayerBulletValid[idx] = 0;
			return;
		}
		isValid = (aNowX[idx] != TheBossX || aNowY[idx] != TheBossY);
		IfPlayerBulletValid[idx] *= isValid;
		if (!isValid)
			PlayerHitBoss[idx] = 1;
		if (D_TheGrid[aNowY[idx] * TheWidth + aNowX[idx]] == 1)
			IfPlayerBulletValid[idx] = 0;
	}
}
void PhyEngS::PrePrepare()
{
	for (auto& i : PlayerHitBoss)
	{
		i = 0;
	}
	for (auto& i : BossHitPlayer)
	{
		i = 0;
	}
	for (int i = 0; i < maxBossIndex; ++i)
	{
		if (IfBossBulletValid[i] == 1)
			TheScreenDraw->Draw(NowX[i], NowY[i], ' ');
	}
	for (int i = 0; i < maxPlayerIndex; ++i)
	{
		if (IfPlayerBulletValid[i] == 1)
			TheScreenDraw->Draw(NowXPlayer[i], NowYPlayer[i], ' ');
	}
}
void PhyEngS::AfterCollision()
{
	abBoss->BossBeHitted();
	abPlayer->PlayerBeHitted();
}
PhyEngS::PhyEngS(std::shared_ptr<Terr> TheNewTerr, int aPlayerX, int aPlayerY, int aBossX, int aBossY)
	:TheTerr(TheNewTerr), maxBossIndex(64), maxPlayerIndex(64), 
	currentBossIndex(0), currentPlayerIndex(0), BossX(aBossX), BossY(aBossY), PlayerX(aPlayerX), PlayerY(aPlayerY)
{
	TheScreenDraw = std::make_shared<ScreenDraw>(TheTerr->GetWidth(), TheTerr->GetHeight(), TheTerr->GetGrid());
	PlayerHitBoss.resize(maxBossIndex);
	BossHitPlayer.resize(maxPlayerIndex);
	speedx.resize(maxBossIndex);
	speedy.resize(maxBossIndex);
	NowX.resize(maxBossIndex);
	NowY.resize(maxBossIndex);
	IfBossBulletValid.resize(maxBossIndex);
	IfPlayerBulletValid.resize(maxPlayerIndex);
	speedxPlayer.resize(maxPlayerIndex);
	speedyPlayer.resize(maxPlayerIndex);
	NowXPlayer.resize(maxPlayerIndex);
	NowYPlayer.resize(maxPlayerIndex);
	for (int i = 0; i < maxBossIndex; ++i)
	{
		IfBossBulletValid[i] = 0;
	}
	for (int i = 0; i < maxPlayerIndex; ++i)
	{
		IfPlayerBulletValid[i] = 0;
	}
	int Width = TheTerr->GetWidth();
	int Height = TheTerr->GetHeight();
	int* TheGrid = new int[Width * Height];
	for (int i = 0; i < Height; ++i)
	{
		for (int j = 0; j < Width; ++j)
		{
			TheGrid[i * Width + j] = (*TheTerr->GetGrid())[i][j];
		}
	}
	hipMalloc((void**)&D_TheGrid, Width * Height * sizeof(int));
	hipMemcpy(D_TheGrid, TheGrid, Width * Height * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&D_speedx, maxBossIndex * sizeof(int));
	hipMalloc((void**)&D_speedy, maxBossIndex * sizeof(int));
	hipMalloc((void**)&D_NowX, maxBossIndex * sizeof(int));
	hipMalloc((void**)&D_NowY, maxBossIndex * sizeof(int));
	hipMalloc((void**)&D_IfBossBulletValid, maxBossIndex * sizeof(int));
	hipMalloc((void**)&D_speedxPlayer, maxPlayerIndex * sizeof(int));
	hipMalloc((void**)&D_speedyPlayer, maxPlayerIndex * sizeof(int));
	hipMalloc((void**)&D_NowXPlayer, maxPlayerIndex * sizeof(int));
	hipMalloc((void**)&D_NowYPlayer, maxPlayerIndex * sizeof(int));
	hipMalloc((void**)&D_IfPlayerBulletValid, maxPlayerIndex * sizeof(int));
	hipMalloc((void**)&D_PlayerHitBoss, maxBossIndex * sizeof(int));
	hipMalloc((void**)&D_BossHitPlayer, maxPlayerIndex * sizeof(int));
	delete[] TheGrid;
}
void PhyEngS::Innitialization(std::shared_ptr<Boss> aBoss, std::shared_ptr<Player> aPlayer)
{
	abBoss = aBoss;
	abPlayer = aPlayer;
}
PhyEngS::~PhyEngS()
{
	hipFree(D_TheGrid);
	hipFree(D_speedx);
	hipFree(D_speedy);
	hipFree(D_NowX);
	hipFree(D_NowY);
	hipFree(D_IfBossBulletValid);
	hipFree(D_speedxPlayer);
	hipFree(D_speedyPlayer);
	hipFree(D_NowXPlayer);
	hipFree(D_NowYPlayer);
	hipFree(D_IfPlayerBulletValid);
	hipFree(D_PlayerHitBoss);
	hipFree(D_BossHitPlayer);
}
void PhyEngS::UpDateBullet()
{
	using namespace std::chrono;
	PrePrepare();
	hipStream_t stream1, stream2, stream3, stream5,stream6;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	hipMemcpyAsync(D_speedx, speedx.data(), maxBossIndex * sizeof(int), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(D_speedxPlayer, speedxPlayer.data(), maxPlayerIndex * sizeof(int), hipMemcpyHostToDevice, stream2);
	hipMemcpyAsync(D_speedy, speedy.data(), maxBossIndex * sizeof(int), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(D_speedyPlayer, speedyPlayer.data(), maxPlayerIndex * sizeof(int), hipMemcpyHostToDevice, stream2);
	hipMemcpyAsync(D_NowX, NowX.data(), maxBossIndex * sizeof(int), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(D_NowXPlayer, NowXPlayer.data(), maxPlayerIndex * sizeof(int), hipMemcpyHostToDevice, stream2);
	hipMemcpyAsync(D_NowY, NowY.data(), maxBossIndex * sizeof(int), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(D_NowYPlayer, NowYPlayer.data(), maxPlayerIndex * sizeof(int), hipMemcpyHostToDevice, stream2);
	hipMemcpyAsync(D_IfBossBulletValid, IfBossBulletValid.data(), maxBossIndex * sizeof(int), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(D_IfPlayerBulletValid, IfPlayerBulletValid.data(), maxPlayerIndex * sizeof(int), hipMemcpyHostToDevice, stream2);
	hipMemcpyAsync(D_PlayerHitBoss, PlayerHitBoss.data(), maxBossIndex * sizeof(int), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(D_BossHitPlayer, BossHitPlayer.data(), maxPlayerIndex * sizeof(int), hipMemcpyHostToDevice, stream2);
	hipStreamSynchronize(stream1);
	hipStreamSynchronize(stream2);
	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);
	hipStreamCreate(&stream3);
	int bossBlockSize = 64;
	int bossNumBlocks = (currentBossIndex + bossBlockSize - 1) / bossBlockSize;
	int playerBlockSize = 64;
	int playerNumBlocks = (currentPlayerIndex + playerBlockSize - 1) / playerBlockSize;
	UpdateBulletKernel << <bossNumBlocks, bossBlockSize, 0, stream3 >> > (D_speedx, D_speedy, D_NowX, D_NowY, D_IfBossBulletValid,
		D_TheGrid, TheTerr->GetWidth(), TheTerr->GetHeight(), maxBossIndex, PlayerX, PlayerY, D_BossHitPlayer);
	hipStreamSynchronize(stream3);
	UpdatePlayerBulletKernel << <playerNumBlocks, playerBlockSize, 0, stream3 >> > (D_speedxPlayer, D_speedyPlayer, D_NowXPlayer, D_NowYPlayer, D_IfPlayerBulletValid,
		D_TheGrid, TheTerr->GetWidth(), TheTerr->GetHeight(), maxPlayerIndex, BossX, BossY, D_PlayerHitBoss, D_NowX, D_NowY, D_IfBossBulletValid, maxBossIndex);
	hipStreamSynchronize(stream3);
	hipStreamDestroy(stream3);
	hipStreamCreate(&stream5);
	hipStreamCreate(&stream6);
	hipMemcpyAsync(speedx.data(), D_speedx, maxBossIndex * sizeof(int), hipMemcpyDeviceToHost, stream5);
	hipMemcpyAsync(speedxPlayer.data(), D_speedxPlayer, maxPlayerIndex * sizeof(int), hipMemcpyDeviceToHost, stream6);
	hipMemcpyAsync(speedy.data(), D_speedy, maxBossIndex * sizeof(int), hipMemcpyDeviceToHost, stream5);
	hipMemcpyAsync(speedyPlayer.data(), D_speedyPlayer, maxPlayerIndex * sizeof(int), hipMemcpyDeviceToHost, stream6);
	hipMemcpyAsync(NowX.data(), D_NowX, maxBossIndex * sizeof(int), hipMemcpyDeviceToHost, stream5);
	hipMemcpyAsync(NowXPlayer.data(), D_NowXPlayer, maxPlayerIndex * sizeof(int), hipMemcpyDeviceToHost, stream6);
	hipMemcpyAsync(NowY.data(), D_NowY, maxBossIndex * sizeof(int), hipMemcpyDeviceToHost, stream5);
	hipMemcpyAsync(NowYPlayer.data(), D_NowYPlayer, maxPlayerIndex * sizeof(int), hipMemcpyDeviceToHost, stream6);
	hipMemcpyAsync(IfBossBulletValid.data(), D_IfBossBulletValid, maxBossIndex * sizeof(int), hipMemcpyDeviceToHost, stream5);
	hipMemcpyAsync(IfPlayerBulletValid.data(), D_IfPlayerBulletValid, maxPlayerIndex * sizeof(int), hipMemcpyDeviceToHost, stream6);
	hipMemcpyAsync(PlayerHitBoss.data(), D_PlayerHitBoss, maxBossIndex * sizeof(int), hipMemcpyDeviceToHost, stream5);
	hipMemcpyAsync(BossHitPlayer.data(), D_BossHitPlayer, maxPlayerIndex * sizeof(int), hipMemcpyDeviceToHost, stream6);
	hipStreamSynchronize(stream5);
	hipStreamSynchronize(stream6);
	hipStreamDestroy(stream5);
	hipStreamDestroy(stream6);
	AfterCollision();
}

void PhyEngS::Draw()
{
	for (int i = 0; i < maxBossIndex; ++i)
	{
		if (IfBossBulletValid[i] == 1)
			TheScreenDraw->Draw(NowX[i], NowY[i], '*');
	}
	for (int i = 0; i < maxPlayerIndex; ++i)
	{
		if (IfPlayerBulletValid[i] == 1)
			TheScreenDraw->Draw(NowXPlayer[i], NowYPlayer[i], 'O');
	}
	TheScreenDraw->Draw(BossX, BossY, BossChar);
	TheScreenDraw->Draw(PlayerX, PlayerY, PlayerChar);
}
int PhyEngS::GetPlayerBeHittedTime()
{
	int j = 0;
	for(int i=0;i<maxBossIndex;++i)
	{
		if (BossHitPlayer[i] == 1)
			++j;
	}
	return j;
}
int PhyEngS::GetBossBeHittedTime()
{
	int j = 0;
	for (int i = 0; i < maxPlayerIndex; ++i)
	{
		if (PlayerHitBoss[i] == 1)
			++j;
	}
	return j;
}
int main()
{
	auto Grid = std::make_shared<Terr>();
	if (Grid->IfCanMove(1, 20) && Grid->IfCanMove(1, 1))
	{
		auto PhyEng = std::make_shared<PhyEngS>(Grid, 1, 20, 1, 1);
		auto Bossphysis = std::make_shared<physis>(PhyEng);
		auto aPlayerphysis = std::make_shared<physis>(PhyEng);
		auto aplayer = std::make_shared<Player>(aPlayerphysis);
		auto aBoss = std::make_shared<Boss>(Bossphysis);
		auto aAINode = std::make_shared<AINode>(aBoss);
		PhyEng->Innitialization(aBoss, aplayer);
		int numnow = 0;
		while (true)
		{
			if (numnow % 3 == 0)
			{
				aAINode->Execute();
			}
			if (numnow % 28 == 1)
				aplayer->PlayerAttack();
			PhyEng->UpDateBullet();
			if (numnow % 180 == 1)
				PhyEng->ReSetBullet();
			PhyEng->Draw();
			PhyEng->TheScreenDraw->Display();
			numnow++;
			if (numnow >= 2048)
				numnow = 0;
			if (aplayer->GetHealth() <= 0)
				break;
			Sleep(100);
		}
	}
}
