#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "PhyEngS.cuh"
#include "physis.h"
#include"Boss.h"
#include"Player.h"
#include"AINode.h"
#include<chrono>
#include<fstream>
#include<thread>
__global__ void UpdateBulletKernel(int* speedx, int* speedy, int* NowX, int* NowY, int* IfBossBulletValid, 
	int* D_TheGrid, int TheWidth, int TheHeight, int currentIndex,const int ThePlayerX,const int ThePlayerY,int*BossHitPlayer,int*PlayerNowX,int *PlayerNowY,int *IfPlayerBulletValid
    ,int maxPlayerIndex)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= currentIndex)
		return;
	if (IfBossBulletValid[idx]==1)
	{
	    NowX[idx] += speedx[idx];
	    NowY[idx] += speedy[idx];
		if (NowX[idx] < 0 || NowX[idx] >= TheWidth || NowY[idx] < 0 || NowY[idx] >= TheHeight)
		{
			IfBossBulletValid[idx] = 0;
			return;
		}
		if (NowX[idx] == ThePlayerX && NowY[idx] == ThePlayerY) {
			BossHitPlayer[idx] = 1;
			IfBossBulletValid[idx] = 0;
			return;
		}
		if (D_TheGrid[NowY[idx] * TheWidth + NowX[idx]] == 1)
		{
			IfBossBulletValid[idx] = 0;
			return;
		}
		for(int i=0;i<maxPlayerIndex;++i)
		{
			if (IfPlayerBulletValid[i] == 1 && NowX[idx] == PlayerNowX[i] && NowY[idx] == PlayerNowY[i])
			{
				IfBossBulletValid[idx] = 0;
				return;
		    }
		}
	}
}
__global__ void UpdatePlayerBulletKernel(int* aspeedx, int* aspeedy, int* aNowX, int* aNowY, int* IfPlayerBulletValid, 
	int* D_TheGrid, int TheWidth, int TheHeight, int currentIndex,const int TheBossX,const int TheBossY,int *PlayerHitBoss)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= currentIndex)
		return;
	if (IfPlayerBulletValid[idx]==1)
	{
		aNowX[idx] += aspeedx[idx];
		aNowY[idx] += aspeedy[idx];
		if (aNowX[idx] < 0 || aNowX[idx] >= TheWidth || aNowY[idx] < 0 || aNowY[idx] >= TheHeight)
		{
			IfPlayerBulletValid[idx] = 0;
			return;
		}
        if(aNowX[idx]==TheBossX&&aNowY[idx]==TheBossY)
		{
			IfPlayerBulletValid[idx] = 0;
			PlayerHitBoss[idx] = 1;
			return;
		}
		if (D_TheGrid[aNowY[idx] * TheWidth + aNowX[idx]] == 1)
			IfPlayerBulletValid[idx] = 0;
	}
}
void PhyEngS::PrePrepare()
{
	DealBossqueue();
	DealPlayerqueue();
	DealBossBullet();
	DealPlayerBullet();
	for (auto& i : PlayerHitBoss)
	{
		i = 0;
	}
	for (auto& i : BossHitPlayer)
	{
		i = 0;
	}
	for (int i = 0; i < maxBossIndex; ++i)
	{
		if (IfBossBulletValid[i] == 1)
			TheScreenDraw->Draw(NowX[i], NowY[i], ' ');
	}
	for (int i = 0; i < maxPlayerIndex; ++i)
	{
		if (IfPlayerBulletValid[i] == 1)
			TheScreenDraw->Draw(NowXPlayer[i], NowYPlayer[i], ' ');
	}
}
void PhyEngS::AfterCollision()
{
	abBoss->BossBeHitted();
	abPlayer->PlayerBeHitted();
}
void PhyEngS::DealBossqueue()
{
	std::lock_guard<std::mutex> Themu(BossMessage);
	if (!Bossqueue.empty())
	{
		auto Top = Bossqueue.front();
		Bossqueue.pop();
		TheScreenDraw->Draw(BossX, BossY, ' ');
		BossX = Top[0];
		BossY = Top[1];
	}
}
void PhyEngS::DealPlayerqueue()
{
	std::lock_guard<std::mutex> Themu(PlayerMessage);
	if(!Playerqueue.empty())
	{
		auto Top = Playerqueue.front();
		Playerqueue.pop();
		TheScreenDraw->Draw(PlayerX, PlayerY, ' ');
		PlayerX = Top[0];
		PlayerY = Top[1];
	}
}
void PhyEngS::DealBossBullet()
{
	BossBulletMu.lock();
	while(!BossBulletqueue.empty())
	{
		auto TheTop = BossBulletqueue.front();
		BossBulletqueue.pop();
		if (IfBossBulletValid[currentBossIndex] == 1)
		{
			currentBossIndex = (currentBossIndex + 1) % maxBossIndex;
			BossBulletMu.unlock();
			return;
		}
		NowX[currentBossIndex] = TheTop[0];
		NowY[currentBossIndex] = TheTop[1];
		speedx[currentBossIndex] = TheTop[2];
		speedy[currentBossIndex] = TheTop[3];
		IfBossBulletValid[currentBossIndex] = 1;
		currentBossIndex = (currentBossIndex + 1)%maxBossIndex;
	}
	BossBulletMu.unlock();
}
void PhyEngS::DealPlayerBullet()
{
	PlayerBulletMu.lock();
	if (PlayerAttackCount != 0)
	{
		PlayerAttackCount = (PlayerAttackCount + 1) %17;
		if (!PlayerBulletqueue.empty())
			PlayerBulletqueue.pop();
		PlayerBulletMu.unlock();
		return;
	}
    if(!PlayerBulletqueue.empty())
	{
		auto TheTop = PlayerBulletqueue.front();
		PlayerBulletqueue.pop();
		if (IfPlayerBulletValid[currentPlayerIndex] == 1) 
		{
			currentPlayerIndex = (currentPlayerIndex + 1) % maxPlayerIndex;
			PlayerBulletMu.unlock();
			return;
		}
		NowXPlayer[currentPlayerIndex] = TheTop[0];
		NowYPlayer[currentPlayerIndex] = TheTop[1];
		speedxPlayer[currentPlayerIndex] = TheTop[2];
		speedyPlayer[currentPlayerIndex] = TheTop[3];
		IfPlayerBulletValid[currentPlayerIndex] = 1;
		currentPlayerIndex = (currentPlayerIndex + 1) % maxPlayerIndex;
		++PlayerAttackCount;
	}
	PlayerBulletMu.unlock();
}
PhyEngS::PhyEngS()
	:maxBossIndex(256), maxPlayerIndex(256), 
	currentBossIndex(0), currentPlayerIndex(0)
{
	TheScreenDraw = std::make_shared<MyScreenDraw>();
	TheTerr = std::make_shared<TerrS>(TheScreenDraw->GetWidth(), TheScreenDraw->GetHeight());
	PlayerX = 0;
	PlayerY = 6;
	BossX = TheScreenDraw->GetWidth() - 1;
	BossY = TheScreenDraw->GetHeight() - 7;
	PlayerHitBoss.resize(maxPlayerIndex);
	BossHitPlayer.resize(maxBossIndex);
	speedx.resize(maxBossIndex);
	speedy.resize(maxBossIndex);
	NowX.resize(maxBossIndex);
	NowY.resize(maxBossIndex);
	IfBossBulletValid.resize(maxBossIndex);
	IfPlayerBulletValid.resize(maxPlayerIndex);
	speedxPlayer.resize(maxPlayerIndex);
	speedyPlayer.resize(maxPlayerIndex);
	NowXPlayer.resize(maxPlayerIndex);
	NowYPlayer.resize(maxPlayerIndex);
	for (int i = 0; i < maxBossIndex; ++i)
	{
		IfBossBulletValid[i] = 0;
	}
	for (int i = 0; i < maxPlayerIndex; ++i)
	{
		IfPlayerBulletValid[i] = 0;
	}
	int Width = TheTerr->GetWidth();
	int Height = TheTerr->GetHeight();
	int* TheGrid = new int[Width * Height];
	for (int i = 0; i < Height; ++i)
	{
		for (int j = 0; j < Width; ++j)
		{
			TheGrid[i * Width + j] = (*TheTerr->GetGrid())[i][j];
		}
	}
	for (int i = 0; i < Height; ++i) {
		for(int j=0;j<Width;++j)
		{
			if (!TheTerr->IfCanMove(j, i))
				TheScreenDraw->Draw(j, i, L'#', 0x0007);
		}
	}
	hipMalloc((void**)&D_TheGrid, Width * Height * sizeof(int));
	hipMemcpy(D_TheGrid, TheGrid, Width * Height * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&D_speedx, maxBossIndex * sizeof(int));
	hipMalloc((void**)&D_speedy, maxBossIndex * sizeof(int));
	hipMalloc((void**)&D_NowX, maxBossIndex * sizeof(int));
	hipMalloc((void**)&D_NowY, maxBossIndex * sizeof(int));
	hipMalloc((void**)&D_IfBossBulletValid, maxBossIndex * sizeof(int));
	hipMalloc((void**)&D_speedxPlayer, maxPlayerIndex * sizeof(int));
	hipMalloc((void**)&D_speedyPlayer, maxPlayerIndex * sizeof(int));
	hipMalloc((void**)&D_NowXPlayer, maxPlayerIndex * sizeof(int));
	hipMalloc((void**)&D_NowYPlayer, maxPlayerIndex * sizeof(int));
	hipMalloc((void**)&D_IfPlayerBulletValid, maxPlayerIndex * sizeof(int));
	hipMalloc((void**)&D_PlayerHitBoss, maxPlayerIndex * sizeof(int));
	hipMalloc((void**)&D_BossHitPlayer, maxBossIndex * sizeof(int));
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	hipStreamCreate(&stream3);
	delete[] TheGrid;
}
void PhyEngS::Innitialization(std::shared_ptr<Boss> aBoss, std::shared_ptr<Player> aPlayer)
{
	abBoss = aBoss;
	abPlayer = aPlayer;
}
PhyEngS::~PhyEngS()
{
	hipFree(D_TheGrid);
	hipFree(D_speedx);
	hipFree(D_speedy);
	hipFree(D_NowX);
	hipFree(D_NowY);
	hipFree(D_IfBossBulletValid);
	hipFree(D_speedxPlayer);
	hipFree(D_speedyPlayer);
	hipFree(D_NowXPlayer);
	hipFree(D_NowYPlayer);
	hipFree(D_IfPlayerBulletValid);
	hipFree(D_PlayerHitBoss);
	hipFree(D_BossHitPlayer);
	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);
	hipStreamDestroy(stream3);
}
void PhyEngS::UpDateBullet()
{
	Tosy->Wait();
	PrePrepare();
	hipMemcpyAsync(D_speedx, speedx.data(), maxBossIndex * sizeof(int), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(D_speedxPlayer, speedxPlayer.data(), maxPlayerIndex * sizeof(int), hipMemcpyHostToDevice, stream2);
	hipMemcpyAsync(D_speedy, speedy.data(), maxBossIndex * sizeof(int), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(D_speedyPlayer, speedyPlayer.data(), maxPlayerIndex * sizeof(int), hipMemcpyHostToDevice, stream2);
	hipMemcpyAsync(D_NowX, NowX.data(), maxBossIndex * sizeof(int), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(D_NowXPlayer, NowXPlayer.data(), maxPlayerIndex * sizeof(int), hipMemcpyHostToDevice, stream2);
	hipMemcpyAsync(D_NowY, NowY.data(), maxBossIndex * sizeof(int), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(D_NowYPlayer, NowYPlayer.data(), maxPlayerIndex * sizeof(int), hipMemcpyHostToDevice, stream2);
	hipMemcpyAsync(D_IfBossBulletValid, IfBossBulletValid.data(), maxBossIndex * sizeof(int), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(D_IfPlayerBulletValid, IfPlayerBulletValid.data(), maxPlayerIndex * sizeof(int), hipMemcpyHostToDevice, stream2);
	hipMemcpyAsync(D_PlayerHitBoss, PlayerHitBoss.data(), maxBossIndex * sizeof(int), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(D_BossHitPlayer, BossHitPlayer.data(), maxPlayerIndex * sizeof(int), hipMemcpyHostToDevice, stream2);
	hipStreamSynchronize(stream1);
	hipStreamSynchronize(stream2);
	int bossBlockSize =256;
	int bossNumBlocks = (maxBossIndex + bossBlockSize - 1) / bossBlockSize;
	int playerBlockSize =256;
	int playerNumBlocks = (maxPlayerIndex + playerBlockSize - 1) / playerBlockSize;
	UpdateBulletKernel << <bossNumBlocks, bossBlockSize, 0, stream3 >> > (D_speedx, D_speedy, D_NowX, D_NowY, D_IfBossBulletValid,
		D_TheGrid, TheTerr->GetWidth(), TheTerr->GetHeight(), maxBossIndex, PlayerX, PlayerY, D_BossHitPlayer,D_NowXPlayer,D_NowYPlayer,D_IfPlayerBulletValid,maxPlayerIndex);
	hipStreamSynchronize(stream3);
	UpdatePlayerBulletKernel << <playerNumBlocks, playerBlockSize, 0, stream3 >> > (D_speedxPlayer, D_speedyPlayer, D_NowXPlayer, D_NowYPlayer, D_IfPlayerBulletValid,
		D_TheGrid, TheTerr->GetWidth(), TheTerr->GetHeight(), maxPlayerIndex, BossX, BossY, D_PlayerHitBoss);
	hipStreamSynchronize(stream3);
	hipMemcpyAsync(speedx.data(), D_speedx, maxBossIndex * sizeof(int), hipMemcpyDeviceToHost, stream1);
	hipMemcpyAsync(speedxPlayer.data(), D_speedxPlayer, maxPlayerIndex * sizeof(int), hipMemcpyDeviceToHost, stream2);
	hipMemcpyAsync(speedy.data(), D_speedy, maxBossIndex * sizeof(int), hipMemcpyDeviceToHost, stream1);
	hipMemcpyAsync(speedyPlayer.data(), D_speedyPlayer, maxPlayerIndex * sizeof(int), hipMemcpyDeviceToHost, stream2);
	hipMemcpyAsync(NowX.data(), D_NowX, maxBossIndex * sizeof(int), hipMemcpyDeviceToHost, stream1);
	hipMemcpyAsync(NowXPlayer.data(), D_NowXPlayer, maxPlayerIndex * sizeof(int), hipMemcpyDeviceToHost, stream2);
	hipMemcpyAsync(NowY.data(), D_NowY, maxBossIndex * sizeof(int), hipMemcpyDeviceToHost, stream1);
	hipMemcpyAsync(NowYPlayer.data(), D_NowYPlayer, maxPlayerIndex * sizeof(int), hipMemcpyDeviceToHost, stream2);
	hipMemcpyAsync(IfBossBulletValid.data(), D_IfBossBulletValid, maxBossIndex * sizeof(int), hipMemcpyDeviceToHost, stream1);
	hipMemcpyAsync(IfPlayerBulletValid.data(), D_IfPlayerBulletValid, maxPlayerIndex * sizeof(int), hipMemcpyDeviceToHost, stream2);
	hipMemcpyAsync(PlayerHitBoss.data(), D_PlayerHitBoss, maxBossIndex * sizeof(int), hipMemcpyDeviceToHost, stream1);
	hipMemcpyAsync(BossHitPlayer.data(), D_BossHitPlayer, maxPlayerIndex * sizeof(int), hipMemcpyDeviceToHost, stream2);
	hipStreamSynchronize(stream1);
	hipStreamSynchronize(stream2);
	AfterCollision();
	Draw();
	TheScreenDraw->Render();
}

void PhyEngS::Draw()
{
	std::lock_guard<std::mutex> abs1(BossBulletMu);
	std::lock_guard<std::mutex> abs2(PlayerBulletMu);
	for (int i = 0; i < maxBossIndex; ++i)
	{
		if (IfBossBulletValid[i] == 1)
			TheScreenDraw->Draw(NowX[i], NowY[i], L'*',0x0004);
	}
	for (int i = 0; i < maxPlayerIndex; ++i)
	{
		if (IfPlayerBulletValid[i] == 1)
			TheScreenDraw->Draw(NowXPlayer[i], NowYPlayer[i], 'o',0x0002);
	}
	TheScreenDraw->Draw(BossX, BossY, BossChar,BossColor);
	TheScreenDraw->Draw(PlayerX, PlayerY, PlayerChar,PlayerColor);
}
int PhyEngS::GetPlayerBeHittedTime()
{
	int j = 0;
	for(int i=0;i<maxBossIndex;++i)
	{
		if (BossHitPlayer[i] == 1)
			++j;
	}
	return j;
}
int PhyEngS::GetBossBeHittedTime()
{
	int j = 0;
	for (int i = 0; i < maxPlayerIndex; ++i)
	{
		if (PlayerHitBoss[i] == 1)
			++j;
	}
	return j;
}
