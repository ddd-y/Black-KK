#include "hip/hip_runtime.h"
#include "PhyEngS.cuh"
#include "hip/hip_runtime.h"
#include ""
#include<chrono>
__global__ void UpdateBulletKernel(int* speedx, int* speedy, int* NowX, int* NowY, bool* IfBossBulletValid, int* D_TheGrid, int TheWidth, int TheHeight, int currentIndex)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < currentIndex)
	{
		if (IfBossBulletValid[idx]==true)
		{
			NowX[idx] += speedx[idx];
			NowY[idx] += speedy[idx];
			if (NowX[idx] < 0 || NowX[idx] >= TheWidth || NowY[idx] < 0 || NowY[idx] >= TheHeight)
			{
				IfBossBulletValid[idx] = false;
			}
			if (D_TheGrid[NowY[idx] * TheWidth + NowX[idx]] == 1)
			{
				IfBossBulletValid[idx] = false;
			}
		}
	}
}
__global__ void UpdatePlayerBulletKernel(int* speedx, int* speedy, int* NowX, int* NowY, bool* IfPlayerBulletValid, int* D_TheGrid, int TheWidth, int TheHeight, int currentIndex)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < currentIndex)
	{
		if (IfPlayerBulletValid[idx])
		{
			NowX[idx] += speedx[idx];
			NowY[idx] += speedy[idx];
			if (D_TheGrid[NowY[idx] * TheWidth + NowX[idx]] == 1)
			{
				IfPlayerBulletValid[idx] = false;
			}
		}
	}
}
PhyEngS::PhyEngS(std::shared_ptr<Terr> TheNewTerr) :TheTerr(TheNewTerr), maxBossIndex(64), maxPlayerIndex(64), currentBossIndex(0), currentPlayerIndex(0)
{
	TheScreenDraw = std::make_shared<ScreenDraw>(TheTerr->GetWidth(), TheTerr->GetHeight(), TheTerr->GetGrid());
	BossBullet.resize(maxBossIndex);
	PlayerBullet.resize(maxPlayerIndex);
	speedx.resize(maxBossIndex);
	speedy.resize(maxBossIndex);
	NowX.resize(maxBossIndex);
	NowY.resize(maxBossIndex);
	IfBossBulletValid.resize(maxBossIndex);
	IfPlayerBulletValid.resize(maxPlayerIndex);
	for (int i = 0; i < maxBossIndex; ++i)
	{
		IfBossBulletValid[i] = false;
	}
	for (int i = 0; i < maxPlayerIndex; ++i)
	{
		IfPlayerBulletValid[i] = false;
	}
	int Width = TheTerr->GetWidth();
	int Height = TheTerr->GetHeight();
	int* TheGrid = new int[Width * Height];
	for (int i = 0; i < Height; ++i)
	{
		for (int j = 0; j < Width; ++j)
		{
			TheGrid[i * Width + j] = (*TheTerr->GetGrid())[i][j];
		}
	}
	hipMalloc((void**)&D_TheGrid, Width * Height * sizeof(int));
	hipMemcpy(D_TheGrid, TheGrid, Width * Height * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&D_speedx, maxBossIndex * sizeof(int));
	hipMalloc((void**)&D_speedy, maxBossIndex * sizeof(int));
	hipMalloc((void**)&D_NowX, maxBossIndex * sizeof(int));
	hipMalloc((void**)&D_NowY, maxBossIndex * sizeof(int));
	hipMalloc((void**)&D_IfBossBulletValid, maxBossIndex * sizeof(bool));
	hipMalloc((void**)&D_speedxPlayer, maxPlayerIndex * sizeof(int));
	hipMalloc((void**)&D_speedyPlayer, maxPlayerIndex * sizeof(int));
	hipMalloc((void**)&D_NowXPlayer, maxPlayerIndex * sizeof(int));
	hipMalloc((void**)&D_NowYPlayer, maxPlayerIndex * sizeof(int));
	hipMalloc((void**)&D_IfPlayerBulletValid, maxPlayerIndex * sizeof(bool));
	delete[] TheGrid;
}
PhyEngS::~PhyEngS()
{
	hipFree(D_TheGrid);
	hipFree(D_speedx);
	hipFree(D_speedy);
	hipFree(D_NowX);
	hipFree(D_NowY);
	hipFree(D_IfBossBulletValid);
	hipFree(D_speedxPlayer);
	hipFree(D_speedyPlayer);
	hipFree(D_NowXPlayer);
	hipFree(D_NowYPlayer);
	hipFree(D_IfPlayerBulletValid);
}
void PhyEngS::UpDateBullet()
{
	for (int i = 0; i < maxBossIndex; ++i)
	{
		if (IfBossBulletValid[i])
			TheScreenDraw->Draw(NowX[i], NowY[i], ' ');
	}
	bool* H_IfPlayerBulletValid = new bool[maxPlayerIndex];
	for (auto i : IfPlayerBulletValid)
	{
		H_IfPlayerBulletValid[i] = i;
	}
	bool* H_IfBossBulletValid = new bool[maxBossIndex];
	for (auto i : IfBossBulletValid)
	{
		H_IfBossBulletValid[i] = i;
	}
	hipStream_t stream1, stream2, stream3, stream4, stream5, stream6;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	hipMemcpyAsync(D_speedx, speedx.data(), maxBossIndex * sizeof(int), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(D_speedxPlayer, speedxPlayer.data(), maxPlayerIndex * sizeof(int), hipMemcpyHostToDevice, stream2);
	hipMemcpyAsync(D_speedy, speedy.data(), maxBossIndex * sizeof(int), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(D_speedyPlayer, speedyPlayer.data(), maxPlayerIndex * sizeof(int), hipMemcpyHostToDevice, stream2);
	hipMemcpyAsync(D_NowX, NowX.data(), maxBossIndex * sizeof(int), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(D_NowXPlayer, NowXPlayer.data(), maxPlayerIndex * sizeof(int), hipMemcpyHostToDevice, stream2);
	hipMemcpyAsync(D_NowY, NowY.data(), maxBossIndex * sizeof(int), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(D_NowYPlayer, NowYPlayer.data(), maxPlayerIndex * sizeof(int), hipMemcpyHostToDevice, stream2);
	hipMemcpyAsync(D_IfBossBulletValid, H_IfBossBulletValid, maxBossIndex * sizeof(bool), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(D_IfPlayerBulletValid, H_IfPlayerBulletValid, maxPlayerIndex * sizeof(bool), hipMemcpyHostToDevice, stream2);
	hipStreamSynchronize(stream1);
	hipStreamSynchronize(stream2);
	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);
	hipStreamCreate(&stream3);
	hipStreamCreate(&stream4);
	int blockSize = 256;
	int numBlocks = (currentBossIndex + blockSize - 1) / blockSize;
	UpdateBulletKernel << <numBlocks, blockSize, 0, stream3 >> > (D_speedx, D_speedy, D_NowX, D_NowY, D_IfBossBulletValid, D_TheGrid, TheTerr->GetWidth(), TheTerr->GetHeight(), currentBossIndex);
	UpdatePlayerBulletKernel << <numBlocks, blockSize, 0, stream4 >> > (D_speedxPlayer, D_speedyPlayer, D_NowXPlayer, D_NowYPlayer, D_IfPlayerBulletValid, D_TheGrid, TheTerr->GetWidth(), TheTerr->GetHeight(), currentPlayerIndex);
	hipStreamSynchronize(stream3);
	hipStreamSynchronize(stream4);
	hipStreamDestroy(stream3);
	hipStreamDestroy(stream4);
	hipStreamCreate(&stream5);
	hipStreamCreate(&stream6);
	hipMemcpyAsync(speedx.data(), D_speedx, maxBossIndex * sizeof(int), hipMemcpyDeviceToHost, stream5);
	hipMemcpyAsync(speedxPlayer.data(), D_speedxPlayer, maxPlayerIndex * sizeof(int), hipMemcpyDeviceToHost, stream6);
	hipMemcpyAsync(speedy.data(), D_speedy, maxBossIndex * sizeof(int), hipMemcpyDeviceToHost, stream5);
	hipMemcpyAsync(speedyPlayer.data(), D_speedyPlayer, maxPlayerIndex * sizeof(int), hipMemcpyDeviceToHost, stream6);
	hipMemcpyAsync(NowX.data(), D_NowX, maxBossIndex * sizeof(int), hipMemcpyDeviceToHost, stream5);
	hipMemcpyAsync(NowXPlayer.data(), D_NowXPlayer, maxPlayerIndex * sizeof(int), hipMemcpyDeviceToHost, stream6);
	hipMemcpyAsync(NowY.data(), D_NowY, maxBossIndex * sizeof(int), hipMemcpyDeviceToHost, stream5);
	hipMemcpyAsync(NowYPlayer.data(), D_NowYPlayer, maxPlayerIndex * sizeof(int), hipMemcpyDeviceToHost, stream6);
	hipMemcpyAsync(H_IfBossBulletValid, D_IfBossBulletValid, maxBossIndex * sizeof(bool), hipMemcpyDeviceToHost, stream5);
	hipMemcpyAsync(H_IfPlayerBulletValid, D_IfPlayerBulletValid, maxPlayerIndex * sizeof(bool), hipMemcpyDeviceToHost, stream6);
	hipStreamSynchronize(stream5);
	hipStreamSynchronize(stream6);
	hipStreamDestroy(stream5);
	hipStreamDestroy(stream6);
	for (int i = 0; i < maxBossIndex; ++i)
	{
		IfBossBulletValid[i] = H_IfBossBulletValid[i];
	}
	for (int i = 0; i < maxPlayerIndex; ++i)
	{
		IfPlayerBulletValid[i] = H_IfPlayerBulletValid[i];
	}
	delete[] H_IfBossBulletValid;
	delete[] H_IfPlayerBulletValid;
	for (int i = 0; i < maxBossIndex; ++i)
	{
		if (IfBossBulletValid[i])
			TheScreenDraw->Draw(NowX[i], NowY[i], 'B');
	}
	TheScreenDraw->Display();
}
int main()
{
	std::shared_ptr<Terr> TheTerr = std::make_shared<Terr>();
	std::shared_ptr<PhyEngS> ThePhyEng = std::make_shared<PhyEngS>(TheTerr);
	ThePhyEng->spawnBossBullet(0, 16, 1, 1);
	ThePhyEng->spawnBossBullet(18, 0, 1, 1);
	ThePhyEng->spawnBossBullet(0, 0, 1, 1);
	ThePhyEng->spawnBossBullet(24, 0, 1, 1);
	ThePhyEng->spawnBossBullet(17, 0, 1, 1);
	while (true)
	{
		ThePhyEng->UpDateBullet();
		Sleep(100);
	}
}